#include "hip/hip_runtime.h"
#include "../../include/gradient/sobel_cuda.cuh"

#include <iostream>

__global__ void helloFromGPU() {
    printf("Hello from the GPU!\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}